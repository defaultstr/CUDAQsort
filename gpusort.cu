#include "gpusort.h"
#include "funcs.cu"

void Init() {
}

void GPUSORT(int size, int *array, int *darray1, int *darray2) {
    bool flip = false;
    int worksize = 1, downsize = 0, paramsize = 0, totsize = size;
    int pivot = workset[0].pivot = (prim_array[0] + prim_array[size/2] + prim_array[size-1])/3;

    while(worksize > 0 && worksize + downsize < MAXSEQ) {
        int blocksize = totsize / MAXSEQ;
        for(int i = 0; i < worksize; i ++) {
            int blockcount = (totsize + blocksize)/blocksize;
            int parent = i, bstart;
            for(int j = 0; j < blockcount; j ++) {
                bstart = workset[i].start + j * blocksize;
                params[paramsize].begin = bstart;
                params[paramsize].end = bstart + blocksize;
                params[paramsize].pivot = pivot;
                params[paramsize].parent = i;
                params[paramsize].last = false;
                paramsize ++;
            }
            params[paramsize-1].end = workset[i].end;
            params[paramsize-1].last = true;
        }
        hipMemcpy(dparams, params, paramsize * sizeof(param), hipMemcpyHostToDevice);
        paramsize = GQSORT<<< TODO >>>(dparams, TODO); 
        hipMemcpy(params, dparams, paramsize * sizeof(param), hipMemcpyDeviceToHost);
        totsize = worksize = 0;
        sequence *temp;
        for(int i = 0; i < paramsize; i ++) {
            if(params[i].begin - params[i].end < size/MAXSEQ) {
                temp = doneset;
                donesize ++;
            } else {
                temp = workset;
                worksize ++;
                totsize += params[i].begin - params[i].end;
            }
            temp[i].begin = params[i].begin;
            temp[i].end = params[i].end;
            temp[i].pivot = params[i].pivot;
        }
    }
    int lqparamsize = 0;
    for(int i = 0; i < worksize; i ++) {
        lqparams[lqparamsize].begin = workset[i].begin;
        lqparams[lqparamsize].end = workset[i].end;
        lqparams[lqparamsize].flip = workset[i].flip;
        lqparams[lqparamsize].sbsize = sbsize;
    }
    hipMemcpy(dlqparams, lqparams, lqparamsize * sizeof(lqparam), hipMemcpyHostToDevice);
    LQSORT<<< TODO >>>(dlqparams, TODO);
    hipMemcpy(lqparams, dlqparams, lqparamsize * sizeof(lqparam), hipMemcpyDeviceToHost);
}
