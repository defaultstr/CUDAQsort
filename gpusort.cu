#include "hip/hip_runtime.h"
#include "gpusort.h"
#include "funcs.cu"

int median(int a, int b, int c) {
    if(a < b) {
        if(b < c) return b;
        else if(a < c) return c;
        else return a;
    } else {
        if(c > a) return a;
        else if(c > b) return c;
        else return b;
    }
}

int Init() {    
    if(hipHostMalloc((void **)&workset, MAXBLOCKS*sizeof(sequence)) != hipSuccess) return -1;
    if(hipHostMalloc((void **)&doneset, MAXBLOCKS*sizeof(sequence)) != hipSuccess) return -2;
    if(hipHostMalloc((void **)&params, MAXBLOCKS*sizeof(param)) != hipSuccess) return -3;
    if(hipHostMalloc((void **)&plist, sizeof(list)) != hipSuccess) return -4;
    if(hipMalloc((void **)&dparams, MAXBLOCKS*sizeof(param)) != hipSuccess) return -5;
    if(hipMalloc((void **)&dplist, sizeof(list)) != hipSuccess) return -6;
    return 0;
}

void GPUSORT(int size, int *array, int *darray1, int *darray2) {
    bool flip = true;
    int worksize = 1, donesize = 0, paramsize = 0, totsize = size;
    int pivot = workset[0].pivot = median(darray1[0], darray1[size/2], darray1[size-1]), index;

    while(worksize > 0 && worksize + donesize < MAXSEQ) {
        int blocksize = totsize / MAXSEQ;
        for(int i = 0; i < worksize; i ++) {
            if(workset[i].end - workset[i].begin < size/MAXSEQ) continue;
            int blockcount = (totsize + blocksize)/blocksize;
            int parent = i, bstart;
            for(int j = 0; j < blockcount; j ++) {
                bstart = workset[i].start + j * blocksize;
                params[paramsize].begin = bstart;
                params[paramsize].end = bstart + blocksize;
                params[paramsize].pivot = workset[i].pivot;
                params[paramsize].parent = i;
                params[paramsize].last = false;
                paramsize ++;
            }
            params[paramsize-1].end = workset[i].end;
            params[paramsize-1].last = true;
        }
        hipMemcpy(dparams, params, paramsize * sizeof(param), hipMemcpyHostToDevice);
        GQSORT1<<< TODO >>>(dparams, TODO); 
        hipMemcpy(plist, dplist, sizeof(list), hipMemcpyDeviceToHost);
		for(int i = 0; i < paramsize; i ++) {
			int l = plist->blockleft[i];
			int r = plist->blockright[i];
			plist->blockleft[i] = workset[params[i].parent].begin;
			plist->blockright[i] = workset[parms[i].parent].end;
			workset[params[i].parent].begin += l;
			workset[params[i].parent].end -= r;
            workset[params[i].parent].maxrpiv = max(workset[params[i].parent].maxrpiv, plist->blockmax[i]);
            workset[params[i].parent].minlpiv = min(workset[params[i].parent].minlpiv, plist->blockmin[i]);
            workset[params[i].parent].maxlpiv = min(workset[params[i].parent].maxlpiv, workset[params[i].parent].pivot);
            workset[params[i].parent].minrpiv = max(workset[params[i].parent].minrpiv, workset[params[i].parent].pivot);
		}
        GQSORT2<<< TODO >>>(dparams, TODO); 
        flip = !flip;
        int oldworksize = worksize, *darray = flip ? darray1 : darray2, b, e;
        totsize = 0, paramsize =0, worksize = 0;
        for(int i = 0; i < oldworksize; i ++) {
            if(workset[i].begin - workset[i].orgbegin < size/MAXSEQ) {
                b = doneset[donesize].begin = workset[i].orgbegin;
                e = doneset[donesize].end = workset[i].begin;
                doneset[donesize].pivot = (workset[i].maxlpiv + workset[i].minlpiv)/2;
                doneset[donesize].flip = flip;
                donesize ++;
            } else {
                totsize += workset[i].begin - workset[i].orgbegin;
                b = params[worksize].begin = workset[i].begin;
                e = params[worksize].end = workset[i].end;
                params[worksize].pivot = (workset[i].maxlpiv + workset[i].minpiv)/2;
                worksize ++;
            }
            if(workset[i].orgend - workset[i].end < size/MAXSEQ) {
                b = doneset[donesize].begin = workset[i].end;
                e = doneset[donesize].end = workset[i].orgend;
                doneset[donesize].pivot = (workset[i].maxlpiv + workset[i].minlpiv)/2;
                doneset[donesize].flip = flip;
                donesize ++;
            } else {
                totsize += workset[i].end - workset[i].orgend;
                b = params[worksize].begin = workset[i].end;
                e = params[worksize].end = workset[i].orgend;
                params[worksize].pivot = (workset[i].maxlpiv + workset[i].minpiv)/2;
                worksize ++;
            }
        }
        for(int i = 0; i < worksize; i ++) {
            workset[i].orgbegin = workset[i].begin = params[i].begin;
            workset[i].orgend = workset[i].end = params[i].begin;
            workset[i].pivot = params[i].pivot;
            workset[i].flip = flip;
        }
    }
    int lqparamsize = 0;
    for(int i = 0; i < worksize; i ++) {
        lqparams[lqparamsize].begin = workset[i].begin;
        lqparams[lqparamsize].end = workset[i].end;
        lqparams[lqparamsize].flip = workset[i].flip;
        lqparams[lqparamsize].sbsize = sbsize;
        lqparamsize ++;
    }
    for(int i = 0; i < donesize; i ++) {
        lqparams[lqparamsize].begin = doneset[i].begin;
        lqparams[lqparamsize].end = doneset[i].end;
        lqparams[lqparamsize].flip = doneset[i].flip;
        lqparams[lqparamsize].sbsize = sbsize;
        lqparamsize ++;
    }
    hipMemcpy(dlqparams, lqparams, lqparamsize * sizeof(lqparam), hipMemcpyHostToDevice);
    LQSORT<<< TODO >>>(dlqparams, TODO);
    hipMemcpy(lqparams, dlqparams, lqparamsize * sizeof(lqparam), hipMemcpyDeviceToHost);
}

void Destroy() {    
    hipHostFree(workset);
    hipHostFree(doneset);
    hipHostFree(params);
    hipFree(dparams);
    hipFree(dplist);
}
