#include "hip/hip_runtime.h"
#include "gpusort.h"

__global__ void
GQSORT1(param *dparams, int *darray1, int *darray2, plist *dplist, bool flip) {
	__shared__ int lt[blockDim.x], gt[blockDim.x];
	int lt, gt;
	int pivot = dparams[blockIdx.x].pivot;
	int* darray = flip ? darray1 : darray2;
	for (int i = dparams[blockIdx.x].start + threadIdx.x;
	     i < dparams[blockIdx.x].end; i += blockDim.x) {
		if (darray[i] < pivot) lt++;
		if (darray[i] >= pivot) gt++;
	}
	lt[threadIdx.x] = lt;
	gt[threadIdx.x] = gt;
	__syncthreads();
	int lsum = 0, gsum = 0;
	if (threadIdx.x == 0) {
		for (int i = 1; i < blockDim.x; i++) {
			lsum += lt[i];
			gsum += gt[i];
			lt[i] = lsum - lt[i];
			gt[i] = gsum - gt[i];
		}
	}
	__syncthreads();
	dplist->left[threadIdx.x + blockIdx.x * blockDim.x] = lt[threadIdx.x];
	dplist->right[threadIdx.x + blockIdx.x * blockDim.x] = gt[threadIdx.x];
	dplist->blockleft[blockIdx.x] = lsum;
	dplist->blockright[blockIdx.x] = gsum;
}

__global__ void
GQSORT2(param *dparams, int *darray1, int *darray2, plist *dplist, bool flip) {
	int* darray = flip ? darray1 : darray2;
	int* darray2 = flip ? darray2 : darray1;
	int lfrom, gfrom;
	lfrom = plist.blockleft[blockIdx.x] + plist.left[threadIdx.x + blockIdx.x * blockDim.x];
	gfrom = plist.blockright[blockIdx.x] - plist.right[threadIdx.x + blockIdx.x * blockDim.x];
	int i = dparams[blockIdx.x].start + threadIdx.x;
	for (; i < dparams[blockIdx.x].end; i += blockDim.x) {
		if (darray[i] < dparams[blockIdx.x].pivot) 
			darray2[lfrom++] = darray[i];
		if (darray[i] >= dparams[blockIdx.x].pivot)
			darray2[gfrom--] = darray[i];
	}
	
	//PART III?

}

