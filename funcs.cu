#include "hip/hip_runtime.h"
#include "gpusort.h"

int max(int a, int b) {
    return a > b ? a : b;
}

int min(int a, int b) {
    return a < b ? a : b;
}

__global__ void
GQSORT1(param *dparams, int *darray1, int *darray2, list *dplist, bool flip) {
	__shared__ int lt[blockDim.x], gt[blockDim.x], minpiv[blockDim.x], maxpiv[blockDim.x];
	int lt, gt, d;
	int pivot = dparams[blockIdx.x].pivot;
	int* darray = flip ? darray1 : darray2; 
    minpiv[threadIdx.x] = darray[dparams[blockIdx.x].start + threadIdx.x];
    maxpiv[threadIdx.x] = darray[dparams[blockIdx.x].start + threadIdx.x];
	for (int i = dparams[blockIdx.x].start + threadIdx.x;
	     i < dparams[blockIdx.x].end; i += blockDim.x) {
        d = darray[i];
		if (d < pivot) lt++;
		if (d > pivot) gt++;
        minpiv[threadIdx.x] = min(minpiv[threadIdx.x], d);
        maxpiv[threadIdx.x] = max(maxpiv[threadIdx.x], d);
	}
	lt[threadIdx.x] = lt;
	gt[threadIdx.x] = gt;
	__syncthreads();
	int lsum = 0, gsum = 0;
    //lt, gt store the end of every thread
	if (threadIdx.x == 0) {
		for (int i = 1; i < blockDim.x; i++) {
			/*lsum += lt[i];
			gsum += gt[i];
			lt[i] = lsum;
			gt[i] = gsum;*/
            lt[i] += lt[i-1];
            gt[i] += gt[i-1];
            minpiv[0] = min(minpiv[0], minpiv[i]);
            maxpiv[0] = max(maxpiv[0], maxpiv[i]);
		}
	}
	__syncthreads();
	dplist->left[threadIdx.x + blockIdx.x * blockDim.x] = lt[threadIdx.x];
	dplist->right[threadIdx.x + blockIdx.x * blockDim.x] = gt[threadIdx.x];
	dplist->blockleft[blockIdx.x] = lt[blockDim.x-1];
	dplist->blockright[blockIdx.x] = gt[blockDim.x-1];
    dplist->blockmin[blockIdx.x] = minpiv[0];
    dplist->blockmax[blockIdx.x] = maxpiv[0];
}

__global__ void
GQSORT2(param *dparams, int *darray1, int *darray2, list *dplist, bool flip) {
    // move the other elements to correct positions
	int* darray = flip ? darray1 : darray2;
	int* darray2 = flip ? darray2 : darray1;
	int lfrom, gfrom;
	lfrom = plist.blockleft[blockIdx.x] + plist.left[threadIdx.x + blockIdx.x * blockDim.x] - 1;
	gfrom = plist.blockright[blockIdx.x] - plist.right[threadIdx.x + blockIdx.x * blockDim.x];
	int i = dparams[blockIdx.x].start + threadIdx.x;
	for (; i < dparams[blockIdx.x].end; i += blockDim.x) {
		if (darray[i] < dparams[blockIdx.x].pivot) 
			darray2[lfrom--] = darray[i];
		if (darray[i] > dparams[blockIdx.x].pivot)
			darray2[gfrom++] = darray[i];
	}

    // fill the pivot
    if(dparams[blockIdx.x].last) {
        int pivot = dparams[blockIdx.x].pivot;
        lfrom = plist.blockleft[blockIdx.x] + plist.left[blockIdx.x * blockDim.x + blockDim.x - 1] + threadIdx.x;
        gfrom = plist.blockright[blockIdx.x] - plist.right[blockIdx.x * blockDim.x + blockDim.x - 1];
        while( lfrom < gfrom) {
            darray2[lfrom] = pivot;
        }
    }

	// return two sequence
    /*if(dparams[blockIdx.x].last) {
        int par = dparams[blockIdx.x].parent;
        dparams[par * 2].begin = 
    }*/
}

