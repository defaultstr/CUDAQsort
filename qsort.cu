#define LENGTH 100000
#define MAX 1000000
#define MAXSEQ 10

void generateRandomArray(long *a, int l) {
    int i;
    srand((unsigned)time(NULL));
    for(i = 0; i < l; i ++)
        a[i] = rand()%MAX + 1;
}

void GPUQSORT(int size, long *prim_array, long *auxi_array) {
    long startpivot = (prim_array[0], prim_array[size/2], prim_array[size-1])/3;
    struct seq_piv
}

//Device Code
__device__ void QuickSortPhase1(long *prim_array, long *auxi_array, int size) {
    //blockDim.x,y,z threadIdx.x,y,z threadIdx.x,y,z
}
__device__ void QuickSortPhase2(long *prim_array, long *auxi_array, int size) {
    //blockDim.x,y,z threadIdx.x,y,z threadIdx.x,y,z
}

int main(int argc, char **argv) {
    size_t size = LENGTH * sizeof(long);
    long host_Array[LENGTH];
    generateRandomArray(host_Array, l);

    long *device_prim_Array;
    long *device_auxi_Array;
    hipMalloc((void**)&device_prim_Array, size);
    hipMalloc((void**)&device_auxi_Array, size);
    hipMemcpy(device_prim_Array, host_Array, size, hipMemcpyHostToDevice);
    //TODO
    int threadsPerBlock = 256;
    int blocksPerGrid = 111;

    hipMemcpy(host_Array, device_prim_Array, size, hipMemcpyDeviceToHost);
    hipFree(device_prim_Array);
    hipFree(device_auxi_Array);
}
