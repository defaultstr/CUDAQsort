#include "gpusort.h"
#define LENGTH 100000
#define MAX 1000000
#define MAXSEQ 10

void generateRandomArray(int *a, int l) {
    int i;
    srand((unsigned)time(NULL));
    for(i = 0; i < l; i ++)
        a[i] = rand()%MAX + 1;
}

int main(int argc, char **argv) {
    size_t size = LENGTH * sizeof(int);
    int host_Array[LENGTH];
    generateRandomArray(host_Array, l);

    int *device_prim_Array;
    int *device_auxi_Array;
    hipMalloc((void**)&device_prim_Array, size);
    hipMalloc((void**)&device_auxi_Array, size);
    hipMemcpy(device_prim_Array, host_Array, size, hipMemcpyHostToDevice);

    Init();
    GPUSORT(host_Array, LENGTH, device_prim_Array, device_auxi_Array);
    Destroy();

    hipMemcpy(host_Array, device_prim_Array, size, hipMemcpyDeviceToHost);
    hipFree(device_prim_Array);
    hipFree(device_auxi_Array);
}
